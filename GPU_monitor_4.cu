#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nvml.h"
#include <iostream>
#include <sys/time.h>
#include <pthread.h>
#include <unistd.h>

using namespace std;

int stop = 0;
nvmlReturn_t mlResult;
nvmlDevice_t *device;
nvmlMemory_t *memory;
nvmlUtilization_t *utilization;
nvmlTemperatureSensors_t sensorType;
char **name, path[256], save_part[128], saveFile[256];
unsigned int *power, *temperature;
int devs, N_run = 1, r_count = 1;
unsigned int nvmlDevs;


void getDate();
void getTime();
void startup(int argc, char **argv);

void *monitora(void *c)
{
   int *tid = (int*)c;


   if(tid[0] == 0)
   {
      sleep(30);
      system(path);
      system("date +%H:%M:%S.%N");
      sleep(30);
      stop = 1;
   }
   else
   {
      FILE *outPtr;
      struct timeval *tvnow;
      tvnow = (struct timeval*)malloc(sizeof(struct timeval)*nvmlDevs);

      sprintf(saveFile, "%s_%04d.dat", save_part, r_count);
      printf("%s\n", saveFile);

      outPtr=fopen(saveFile,"w");
      if(outPtr==NULL)
      {
         printf("Falha na abertura");
         pthread_exit(c);
      }
      fprintf(outPtr, "|      Time       | Device name | Device # | Memory unit |   Free   |   Used   |   Total   | Util. Rate Memory/GPU | Power (W) | Temperature (C) |\n" );
      while(!stop)
      {
         for(int i = 0; i < nvmlDevs; i++)
         {
            gettimeofday(&tvnow[i], NULL);
            mlResult = nvmlDeviceGetName(device[i], name[i], 50);
            mlResult = nvmlDeviceGetMemoryInfo(device[i], &memory[i]);
            mlResult = nvmlDeviceGetPowerUsage(device[i], &power[i]);
/*            if(NVML_SUCCESS != mlResult)
            {
               fprintf(outPtr, "Failed to get Power readings: %s\n", nvmlErrorString(mlResult));
            }
*/
            mlResult = nvmlDeviceGetTemperature(device[i], sensorType, &temperature[i]);
/*            if(NVML_SUCCESS != mlResult)
           {
              fprintf(outPtr, "Failed to get temperture: %s\n", nvmlErrorString(mlResult));
           }
*/
           mlResult =  nvmlDeviceGetUtilizationRates(device[i], &utilization[i]);
           if(NVML_SUCCESS != mlResult)
           {
              fprintf(outPtr, "Failed to get utilization rates: %s\n", nvmlErrorString(mlResult));
           }

        }

//	fprintf(outPtr, "| Time | Device name | Device # | Free memory | Used memory | Total memory | Power (W) | Temperature (C) |\n" );
        for(int i = 0; i < nvmlDevs; i++)
        {
           struct tm* tm = localtime(&tvnow[i].tv_sec);
	   fprintf(outPtr, "|%3d:%02d:%02d.%06ld | %s | %5d    |%8s     |%9.2Lf |%9.2Lf |%10.2Lf |%9i%% |%9i%% |%9.3f  |%10d       |\n", tm->tm_hour, tm->tm_min, tm->tm_sec, tvnow[i].tv_usec, name[i], i, "MB",(long double)memory[i].free/1048576.0,(long double)memory[i].used/1048576.0, (long double)memory[i].total/1048576.0, utilization[i].memory, utilization[i].gpu , (float)power[i]/1000.0f, temperature[i]);


           /*fprintf(outPtr, "%d:%02d:%02d.%06ld\n", tm->tm_hour, tm->tm_min, tm->tm_sec, tvnow[i].tv_usec);
           fprintf(outPtr, "nvml Device Name: %s\n", name[i]);
           fprintf(outPtr, "device %d memory: free %.2LfMB, used %.2LfMB, Total %.2LfMB\n", i, (long double)memory[i].free/1048576.0,(long double)memory[i].used/1048576.0, (long double)memory[i].total/1048576.0);
           fprintf(outPtr, "Power usage: %d watts. Tempereture: %d C\n", power[i], temperature[i]);
           */
        }

     }

     fclose(outPtr);

   }

   pthread_exit(c);
}


void getTime()
{
   struct timeval tvnow;
   gettimeofday(&tvnow, NULL);
   struct tm* tm = localtime(&tvnow.tv_sec);

   printf("%d:%02d:%02d.%06ld\n", tm->tm_hour, tm->tm_min, tm->tm_sec, tvnow.tv_usec);

}

int main(int argc, char **argv)
{
   mlResult = nvmlInit();
   if(NVML_SUCCESS != mlResult)
      printf("Failed to Initialize NVML: %s\n", nvmlErrorString(mlResult));

   startup(argc, argv);

   nvmlDeviceGetCount(&nvmlDevs);

   printf("cuda devs = %d, nvml devs = %d\n", devs, nvmlDevs);

//   path = (char*)malloc(sizeof(char)*256);
   device = (nvmlDevice_t *)malloc(sizeof(nvmlDevice_t)*nvmlDevs);
   memory = (nvmlMemory_t*)malloc(sizeof(nvmlMemory_t)*nvmlDevs);
   name = (char**)malloc(sizeof(char*)*nvmlDevs);
   power = (unsigned int*)malloc(sizeof(unsigned int)*nvmlDevs);
   temperature = (unsigned int*)malloc(sizeof(unsigned int)*nvmlDevs);
   utilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t)*nvmlDevs);

/*   path[0] = '\0';
   strcat(path, "date +%H:%M:%S.%N && ./");
   strcat(path, argv[1]);
   printf("path = %s\n", path);
*/

   for(int i = 0; i < nvmlDevs; i++)
   {
      mlResult = nvmlDeviceGetHandleByIndex(i, &device[i]);
      name[i] = (char*)malloc(sizeof(char)*50);
   } 

   for(int i = 0; i < N_run; i++)
   {
      stop = 0;
      r_count = i+1;
      int tidx[2];
      pthread_t tid[2]; 
      printf("run %d\n", i); 

      for(int i = 0; i < 2; i++)
      {
         tidx[i] = i;
         pthread_create(&tid[i], NULL, &monitora, (void*)&tidx[i]);
      }
      for(int k = 0; k < 2; k++)
      {
         pthread_join(tid[k], NULL);
      }
   }   

   mlResult = nvmlShutdown();
   if(NVML_SUCCESS != mlResult)
   {
      printf("Failed to shutdown NVML: %s\n", nvmlErrorString(mlResult));

      printf("Press ENTER to continue...\n");
      getchar();
   }


   return 0;
}

void startup(int argc, char **argv)
{
   int i;

   for(i = 0; i < argc; i++)
   {    
      printf("%s\n", argv[i]);
   } 

   if(argc > 1)
   {
      for(i = 1; i < argc; i++)
      {
         if ((!strcmp(argv[i], "-p")) || (!strcmp(argv[i], "-P")))
         {  
            if(argv[i+1][0] != '-')
            {
               //path[0] = '\0';
               strcpy(path, "date +%H:%M:%S.%N && ./");
               strcat(path, argv[i+1]);
               printf("path = %s\n", path);
               i++;
            }
            else
            {
               printf("Parameter \"-p\"/\"-P\" must be followed by the path of the application to be evaluated\n");
               printf("current line %d, path = %s\n", __LINE__, path);
               exit(0);
            }
         }
         else if((!strcmp(argv[i], "-f")) || (!strcmp(argv[i], "--file")))
         {
            if(argv[i+1][0] != '-')
            {
               strcpy(save_part, argv[i+1]);
               printf("save_part = %s\n", save_part);
               i++;
            }
            else
            {
               printf("Parameter \"--file\"/\"-f\" must be followed by the result output file name!\n");
               printf("current line %d, save_part = %s\n", __LINE__, save_part);
               exit(0);
            }
         }
         else if((!strcmp(argv[i], "--round")) || (!strcmp(argv[i], "-r")) )
         {
            if(atoi(argv[i+1]) >= 1)
            {
               N_run = atoi(argv[i+1]);
               i = i+1;
            }
            else
            {
               printf("Parameter \"--round\"/\"-r\" must be followed by a positive integer !\n");
               exit(0);
            }
         }
         else
         {
            printf("TSPd_GA : Invalid option! \"%s\", try \'TSPd_GA --help\' for more information.\n", argv[i]);
            exit(0);
         }
      }
   }

   for(i = 1; i <= N_run; i++)
   {
      //strcpy(saveFile, save_part);
      sprintf(saveFile, "%s_%04d.dat", save_part, i);
      printf("%s\n", saveFile);
   }

}

